#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

//Working group code
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "Kernel.h"

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

// Trig constants:
#define PI (3.1416)

//#define BETA (0.1)
// Right angle constants:
#define THETA_R (10.0 / 180.0 * PI) // 5 degrees

// Sampling constants:
#define S_SIGMA_P (0.8)
#define S_SIGMA_T (15.0 / 90.0 * PI)

//In the original implementation, number of threads in a group was set to the WARP size, which we can do with 32
#define WARP_SIZE 32


struct vertex
{
	double x;
	double y;
	double z;
};

struct rectangle
{
	int point1Index;
	int point2Index;
	int point3Index;
	int point4Index;
	int SourceIndex;
};

struct positionAndRotation
{
	double x;
	double y;
	double z;

	double rotX;
	double rotY;
	double rotZ;
	bool frozen;

	double length;
	double width;
};

struct targetRangeStruct {
	double targetRangeStart;
	double targetRangeEnd;
};

struct relationshipStruct
{
	targetRangeStruct TargetRange;
	targetRangeStruct AngleRange;
	int SourceIndex;
	int TargetIndex;
	double DegreesOfAtrraction;
};

struct Surface
{
	int nObjs;
	int nRelationships;
	int nClearances;

	// Weights
	float WeightFocalPoint;
	float WeightPairWise;
	float WeightVisualBalance;
	float WeightSymmetry;
	float WeightOffLimits;
	float WeightClearance;
	float WeightSurfaceArea;

	// Centroid
	double centroidX;
	double centroidY;

	// Focal point
	double focalX;
	double focalY;
	double focalRot;
};

struct gpuConfig
{
	int gridxDim;
	int gridyDim;
	int blockxDim;
	int blockyDim;
	int blockzDim;
	int iterations;
};

struct point
{
	float x, y, z, rotX, rotY, rotZ;
	bool frozen;

	double length;
	double width;
};

struct resultCosts
{
	float totalCosts;
	float PairWiseCosts;
	float VisualBalanceCosts;
	float FocalPointCosts;
	float SymmetryCosts;
	float ClearanceCosts;
	float OffLimitsCosts;
	float SurfaceAreaCosts;
};

struct result {
	point *points;
	resultCosts costs;
};

__global__ void initRNG(hiprandState *const rngStates, const unsigned int seed)
{
	// Determine thread ID
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("tid: %d\n", tid);
	//printf("seed: %d\n", seed);
	// Initialise the RNG
	hiprand_init(seed + tid, tid, 0, &rngStates[tid]);
}

__device__ double Distance(float xi, float yi, float xj, float yj) 
{
	double dX = xi - xj;
	double dY = yi - yj;
	return sqrt(dX * dX + dY * dY);
}

//Determines the angular difference between two objects where i is oriented to j (i is bearing to j)
__device__ double theta(float xi, float yi, float xj, float yj, float ti) {
	double dX = xi - xj;
	double dY = yi - yj;
	double theta_p = atan2(dY, dX); //gives us the angle between -PI and PI

									//and now between 0 and 2*pi
	theta_p = (theta_p < 0) ? 2 * PI + theta_p : theta_p;
	//printf("theta_p=%f,ti=%f\n",theta_p,ti);
	//return the re-oriented angle
	double theta = theta_p - ti;
	return (theta < 0) ? 2 * PI + theta : theta;

}

// Tj is the rotation
__device__ float phi(float xi, float yi, float xj, float yj, float tj)
{
	return atan2(yi - yj, xi - xj) - tj + PI / 2.0;
}

//To get coop groups working, we need to remove the useage of shared memory. We can do this using a shuffle/reduce
//taken from https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
template<int tile_sz>
 __device__ float reduce(cg::thread_block_tile<tile_sz> group, float val) { //Group size is static, so no need to include that
	for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
		val += group.shfl_down(val, offset);
	}
	return val;
}

//Modest reduction algorithm with a lot of room to improve upon
/*__device__ void reduce(cg::thread_group group,float *values, int n) { //Size of the array (from how we use it, it's at most of size blockDim.x)
	int stride = group.size()/2;
	int tid = group.thread_rank();
	int size = n;
	//We make the very important for parallel reduction assumptions that blockDim is a power of two and values is a multiple of blockdim
	//We can do this because we control those
	while (size > 1) {
		for (int i = tid + stride; i < size; i += stride) {
			values[tid] += values[i];
			//printf("tid = %d with value %f\n", tid, values);
		}
		size = size / 2;
		stride = stride / 2;
		group.sync();
		//local variable per thread, so no race condition
	}
}*/
template<int tile_sz>
__device__ double VisualBalanceCosts(cg::thread_block_tile<tile_sz> group, Surface *srf, positionAndRotation *cfg)
{
	int tid =  group.thread_rank();
	int step = group.size();
	float nx;// = 0;
	float ny;// = 0;
	float denom;// = 0;
	//because of multiple share blocks, we do an atomic add instead of the reduce method
	nx = 0.0;
	ny = 0.0;
	denom = 0.0;
	for (int i = tid; i < srf->nObjs; i+=step)
	{
		float area = cfg[i].length * cfg[i].width;
		nx += area * cfg[i].x;
		ny += area * cfg[i].y;
		denom += area;
	}
	group.sync();
	reduce<tile_sz>(group, nx);
	reduce<tile_sz>(group, ny);
	reduce<tile_sz>(group, denom);
	// Distance between all summed areas and points divided by the areas and the room's centroid
	return  Distance(nx / denom, ny / denom, srf->centroidX / 2, srf->centroidY / 2); //Because we are all reducing, all values should be the same
}

template<int tile_sz>
__device__ double PairWiseCosts(cg::thread_block_tile<tile_sz> group,Surface *srf, positionAndRotation* cfg, relationshipStruct *rs)
{
	int tid = group.thread_rank();
	int step = group.size();
	float values;
	values = 0.0f; //Since it's size blockDim, we can have each of them treat it as the starting value
	for (int i = 0; i < srf->nRelationships; i+=step)
	{
		// Look up source index from relationship and retrieve object using that index.
		double distance = Distance(cfg[rs[i].SourceIndex].x, cfg[rs[i].SourceIndex].y, cfg[rs[i].TargetIndex].x, cfg[rs[i].TargetIndex].y);
		//printf("Distance: %f Range start: %f Range end: %f\n", distance, rs[i].TargetRange.targetRangeStart, rs[i].TargetRange.targetRangeEnd);
		//penalize if we are too close
		if (distance < rs[i].TargetRange.targetRangeStart)
		{
			double fraction = distance / rs[i].TargetRange.targetRangeStart;
			values -= (fraction * fraction);
		}
		//penalize if we are too far
		else if (distance > rs[i].TargetRange.targetRangeEnd)
		{
			double fraction = rs[i].TargetRange.targetRangeEnd / distance;
			values -= (fraction * fraction);
		}
		else {
			values -= 1;
		}
		// Else don't do anything as 0 indicates a perfect solution
	}
	group.sync();
	reduce<tile_sz>(group, values);
	//printf("Clearance costs error: %f\n", error);
	return (double)values;
}

//This functional principle uses a lookup (relationshipStruct) to determine weights from a recommended angle
//This is not the facing angle but the distance angle (so, the target rotated around the source)
template<int tile_sz>
__device__ double PairWiseTotalCosts(cg::thread_block_tile<tile_sz> group, Surface *srf, positionAndRotation* cfg, relationshipStruct *rs)
{
	int tid = group.thread_rank();
	int step = group.size();
	float values;
	values = 0.0f; //Since it's size blockDim, we can have each of them treat it as the starting value
						//assuming (0,2*PI]
	for (int i = tid; i < srf->nRelationships; i += step)
	{
		// We use phi to calculate the angle between the rotation of the object and the target object
		double distance = Distance(cfg[rs[i].SourceIndex].x, cfg[rs[i].SourceIndex].y, cfg[rs[i].TargetIndex].x, cfg[rs[i].TargetIndex].y);
		double angle = theta(cfg[rs[i].SourceIndex].x, cfg[rs[i].SourceIndex].y, cfg[rs[i].TargetIndex].x, cfg[rs[i].TargetIndex].y, cfg[rs[i].TargetIndex].rotY);
		
		//Score distance calculation
		double score = (distance < rs[i].TargetRange.targetRangeStart) ? powf(distance / rs[i].TargetRange.targetRangeStart, rs[i].DegreesOfAtrraction) : 1.0;
		score        = (distance > rs[i].TargetRange.targetRangeEnd)   ? powf(rs[i].TargetRange.targetRangeEnd / distance  , rs[i].DegreesOfAtrraction)  : 1.0;

		//For now, we assume start is greater than end
		double norm    = (rs[i].TargetRange.targetRangeStart < rs[i].TargetRange.targetRangeEnd)? rs[i].AngleRange.targetRangeEnd - rs[i].AngleRange.targetRangeStart : 
																								  rs[i].AngleRange.targetRangeStart - rs[i].AngleRange.targetRangeEnd; //The max distance away is half the slice that is in the no zone 
		norm = (2.0 * PI - norm) / 2.0;
		double a_score = (rs[i].AngleRange.targetRangeEnd < angle || angle < rs[i].AngleRange.targetRangeEnd) ? fmin(fabs(distance - rs[i].AngleRange.targetRangeStart), 
																													 fabs(distance - rs[i].AngleRange.targetRangeEnd)) / norm : 1.0;
		values -= score*a_score; //So, best score we can do is -1, and everything else degrades from there
	}
	group.sync();
	reduce<tile_sz>(group, values);
	//printf("Clearance costs error: %f\n", error);
	return values;
}

template<int tile_sz>
__device__ double FocalPointCosts(cg::thread_block_tile<tile_sz> group, Surface *srf, positionAndRotation* cfg)
{
	int tid = group.thread_rank();
	int step = group.size();
	float values;
	values = 0.0f; //Since it's size blockDim, we can have each of them treat it as the starting value
	for (int i = tid; i < srf->nObjs; i += step)
	{
		float phi_fi = phi(srf->focalX, srf->focalY, cfg[i].x, cfg[i].y, cfg[i].rotY);
		// Old implementation of grouping, all objects that belong to the seat category are used in the focal point calculation
		// For now we default to all objects, focal point grouping will come later
		//int s_i = s(r.c[i]);

		// sum += s_i * cos(phi_fi);
		values -= cos(phi_fi);
	}
	group.sync();
	reduce<tile_sz>(group, values);
	//printf("tid = %d, value = %f\n", tid, values[tid]);
	//printf("Clearance costs error: %f\n", error);
	return (double)values;
}

template<int tile_sz>
__device__ float SymmetryCosts(cg::thread_block_tile<tile_sz> group, Surface *srf, positionAndRotation* cfg)
{
	int tid = group.thread_rank();
	int step = group.size();
	float values;
	values = 0.0f; //Since it's size blockDim, we can have each of them treat it as the starting value
	for (int i = tid; i < srf->nObjs; i += step)
	{
		float maxVal = 0;

		float ux = cos(srf->focalRot);
		float uy = sin(srf->focalRot);
		float s = 2 * (srf->focalX * ux + srf->focalY * uy - (cfg[i].x * ux + cfg[i].y * uy));  // s = 2 * (f * u - v * u)

																								// r is the reflection of g across the symmetry axis defined by p.
		float rx_i = cfg[i].x + s * ux;
		float ry_i = cfg[i].y + s * uy;
		float rRot_i = 2 * srf->focalRot - cfg[i].rotY;
		if (rRot_i < -PI)
			rRot_i += 2 * PI;

		for (int j = 0; j < srf->nObjs; j++)
		{
			// Types should be the same, this probably works great with their limited amount of types but will probably not work that great for us. Perhaps define a group?
			int gamma_ij = 1;
			float dp = Distance(cfg[j].x, cfg[j].y, rx_i, ry_i);
			float dt = cfg[j].rotY - rRot_i;
			if (dt > PI)
				dt -= 2 * PI;

			float val = gamma_ij * (5 - sqrt(dp) - 0.4 * fabs(dt));
			maxVal = fmaxf(maxVal, val);
		}

		values -= maxVal;
	}

	group.sync();
	reduce<tile_sz>(group, values);
	//printf("Clearance costs error: %f\n", error);
	return values;
}


__device__ float calculateIntersectionArea(vertex rect1Min, vertex rect1Max, vertex rect2Min, vertex rect2Max) {
	// printf("Clearance rectangle 1: Min X: %f Y: %f Max X: %f Y: %f\n", rect1Min.x, rect1Min.y, rect1Max.x, rect1Max.y);
	// printf("Clearance rectangle 2: Min X: %f Y: %f Max X: %f Y: %f\n", rect2Min.x, rect2Min.y, rect2Max.x, rect2Max.y);
	// for each two rectangles, find out their intersection. Increase the error using the area
	float x5 = fmaxf(rect1Min.x, rect2Min.x);
	float y5 = fmaxf(rect1Min.y, rect2Min.y);
	float x6 = fminf(rect1Max.x, rect2Max.x);
	float y6 = fminf(rect1Max.y, rect2Max.y);

	// Check if proper rectangle, if so it is an intersection.
	if (x5 >= x6 || y5 >= y6)
		return 0.0f;

	// printf("Intersection rectangle: Min X: %f Y: %f Max X: %f Y: %f\n", x5, y5, x6, y6);

	// Calculate area and add to error
	float area = (x6 - x5) * (y6 - y5);
	// printf("Area intersection rectangle: %f\n", area);
	return area;
}

__device__ void createComplementRectangle(vertex srfRectMin, vertex srfRectMax, vertex *complementRectangle1, vertex *complementRectangle2, vertex *complementRectangle3, vertex *complementRectangle4) {
	// 0 is min value, 1 is max value
	complementRectangle1[0].x = -DBL_MAX;
	complementRectangle1[0].y = -DBL_MAX;
	complementRectangle1[1].x = DBL_MAX;
	complementRectangle1[1].y = srfRectMin.y;

	complementRectangle2[0].x = -DBL_MAX;
	complementRectangle2[0].y = srfRectMin.y;
	complementRectangle2[1].x = srfRectMin.x;
	complementRectangle2[1].y = srfRectMax.y;

	complementRectangle3[0].x = -DBL_MAX;
	complementRectangle3[0].y = srfRectMax.y;
	complementRectangle3[1].x = DBL_MAX;
	complementRectangle3[1].y = DBL_MAX;

	complementRectangle4[0].x = srfRectMax.x;
	complementRectangle4[0].y = srfRectMin.y;
	complementRectangle4[1].x = DBL_MAX;
	complementRectangle4[1].y = srfRectMax.y;
}

__device__ vertex minValue(vertex *vertices, int startIndexVertices, float xtranslation, float ytranslation) {
	vertex rect1;
	rect1.x = DBL_MAX;
	rect1.y = DBL_MAX;
	rect1.z = 0;
	rect1.x = (rect1.x > vertices[startIndexVertices].x + xtranslation) ? vertices[startIndexVertices].x : rect1.x;
	rect1.x = (rect1.x > vertices[startIndexVertices + 1].x + xtranslation) ? vertices[startIndexVertices + 1].x + xtranslation : rect1.x;
	rect1.x = (rect1.x > vertices[startIndexVertices + 2].x + xtranslation) ? vertices[startIndexVertices + 2].x + xtranslation : rect1.x;
	rect1.x = (rect1.x > vertices[startIndexVertices + 3].x + xtranslation) ? vertices[startIndexVertices + 3].x + xtranslation : rect1.x;

	rect1.y = (rect1.y > vertices[startIndexVertices].y + ytranslation) ? vertices[startIndexVertices].y + ytranslation : rect1.y;
	rect1.y = (rect1.y > vertices[startIndexVertices + 1].y + ytranslation) ? vertices[startIndexVertices + 1].y + ytranslation : rect1.y;
	rect1.y = (rect1.y > vertices[startIndexVertices + 2].y + ytranslation) ? vertices[startIndexVertices + 2].y + ytranslation : rect1.y;
	rect1.y = (rect1.y > vertices[startIndexVertices + 3].y + ytranslation) ? vertices[startIndexVertices + 3].y + ytranslation : rect1.y;
	//printf("Min value vector after translation: X: %f Y: %f\n", rect1.x, rect1.y);
	return rect1;
}

__device__ vertex maxValue(vertex *vertices, int startIndexVertices, float xtranslation, float ytranslation) {
	vertex rect1;
	rect1.x = -DBL_MAX;
	rect1.y = -DBL_MAX;
	rect1.z = 0;

	rect1.x = (rect1.x < vertices[startIndexVertices].x + xtranslation) ? vertices[startIndexVertices].x + xtranslation : rect1.x;
	rect1.x = (rect1.x < vertices[startIndexVertices + 1].x + xtranslation) ? vertices[startIndexVertices + 1].x + xtranslation : rect1.x;
	rect1.x = (rect1.x < vertices[startIndexVertices + 2].x + xtranslation) ? vertices[startIndexVertices + 2].x + xtranslation : rect1.x;
	rect1.x = (rect1.x < vertices[startIndexVertices + 3].x + xtranslation) ? vertices[startIndexVertices + 3].x + xtranslation : rect1.x;

	rect1.y = (rect1.y < vertices[startIndexVertices].y + ytranslation) ? vertices[startIndexVertices].y + ytranslation : rect1.y;
	rect1.y = (rect1.y < vertices[startIndexVertices + 1].y + ytranslation) ? vertices[startIndexVertices + 1].y + ytranslation : rect1.y;
	rect1.y = (rect1.y < vertices[startIndexVertices + 2].y + ytranslation) ? vertices[startIndexVertices + 2].y + ytranslation : rect1.y;
	rect1.y = (rect1.y < vertices[startIndexVertices + 3].y + ytranslation) ? vertices[startIndexVertices + 3].y + ytranslation : rect1.y;
	//printf("Max value vector after translation: X: %f Y: %f\n", rect1.x, rect1.y);
	return rect1;
}

// Clearance costs is calculated by determining any intersections between clearances and offlimits. Clearances may overlap with other clearances
template<int tile_sz>
__device__ float ClearanceCosts(cg::thread_block_tile<tile_sz> group, Surface *srf, positionAndRotation* cfg, vertex *vertices, rectangle *clearances, rectangle *offlimits)
{
	int tid = group.thread_rank();
	int step = group.size();
	float values;
	values = 0.0f; //Since it's size blockDim, we can have each of them treat it as the starting value
	for (int i = tid; i < srf->nClearances; i+=step) {
		vertex rect1Min = minValue(vertices, clearances[i].point1Index, cfg[clearances[i].SourceIndex].x, cfg[clearances[i].SourceIndex].y);
		vertex rect1Max = maxValue(vertices, clearances[i].point1Index, cfg[clearances[i].SourceIndex].x, cfg[clearances[i].SourceIndex].y);
		for (int j = tid; j < srf->nObjs; j += blockDim.x) {
			vertex rect2Min = minValue(vertices, offlimits[j].point1Index, cfg[j].x, cfg[j].y);
			vertex rect2Max = maxValue(vertices, offlimits[j].point1Index, cfg[j].x, cfg[j].y);
			// Determine max and min vectors of clearance rectangles
			// rectangle #1
			//printf("Clearance\n");
			//printf("Translation: X: %f Y: %f\n", cfg[i].x, cfg[i].y);
			// printf("Clearance rectangle %d: Min X: %f Y: %f Max X: %f Y: %f\n", i, rect1Min.x, rect1Min.y, rect1Max.x, rect1Max.y);
			// rectangle #2
			//printf("Off limits\n");
			//printf("Translation: X: %f Y: %f\n", cfg[j].x, cfg[j].y);
			// printf("Clearance rectangle %d: Min X: %f Y: %f Max X: %f Y: %f\n", i, rect2Min.x, rect2Min.y, rect2Max.x, rect2Max.y);
			float area = calculateIntersectionArea(rect1Min, rect1Max, rect2Min, rect2Max);
			//printf("Area intersection rectangle %d and %d: %f\n", i, j, area);
			values = area; //Clearence penalty should be positive
		}
	}
	group.sync();
	reduce<tile_sz>(group, values);
	//printf("Clearance costs error: %f\n", error);
	return values;
}

// Both clearance as offlimits may not lie outside of the surface area
template<int tile_sz>
__device__ float SurfaceAreaCosts(cg::thread_block_tile<tile_sz> group, Surface *srf, positionAndRotation* cfg, vertex *vertices, rectangle *clearances, rectangle *offlimits, vertex *surfaceRectangle) {
	//printf("Surface cost calculation\n");

	int tid = group.thread_rank();
	int step = group.size();
	float values;
	values = 0.0f; //Since it's size blockDim, we can have each of them treat it as the starting value
	// Describe the complement of surfaceRectangle as four rectangles (using their min and max values)
	vertex complementRectangle1[2];
	vertex complementRectangle2[2];
	vertex complementRectangle3[2];
	vertex complementRectangle4[2];

	// Figure out min and max vectors of surface rectangle
	vertex srfRect1Min = minValue(surfaceRectangle, 0, 0, 0);
	vertex srfRect1Max = maxValue(surfaceRectangle, 0, 0, 0);

	//This gives us the total rectangle outside our surface area
	createComplementRectangle(srfRect1Min, srfRect1Max, complementRectangle1, complementRectangle2, complementRectangle3, complementRectangle4);

	for (int i = tid; i < srf->nClearances; i += step) {
		// Determine max and min vectors of clearance rectangles
		// rectangle #1
		//Old way of doing things through a memory error
		//vertex rect1Min = minValue(vertices, clearances[i].point1Index, cfg[i].x, cfg[i].y);
		//vertex rect1Max = maxValue(vertices, clearances[i].point1Index, cfg[i].x, cfg[i].y);
		vertex rect1Min = minValue(vertices, clearances[i].point1Index, cfg[clearances[i].SourceIndex].x, cfg[clearances[i].SourceIndex].y);
		vertex rect1Max = maxValue(vertices, clearances[i].point1Index, cfg[clearances[i].SourceIndex].x, cfg[clearances[i].SourceIndex].y);

		// printf("Clearance rectangle %d: Min X: %f Y: %f Max X: %f Y: %f\n", i, rect1Min.x, rect1Min.y, rect1Max.x, rect1Max.y);


		// printf("Area intersection rectangle %d and %d: %f\n", i, j, area);
		values -= calculateIntersectionArea(rect1Min, rect1Max, complementRectangle1[0], complementRectangle1[1]);
		values -= calculateIntersectionArea(rect1Min, rect1Max, complementRectangle2[0], complementRectangle2[1]);
		values -= calculateIntersectionArea(rect1Min, rect1Max, complementRectangle3[0], complementRectangle3[1]);
		values -= calculateIntersectionArea(rect1Min, rect1Max, complementRectangle4[0], complementRectangle4[1]);
	}

	for (int j = tid; j < srf->nObjs; j += step) {
		// Determine max and min vectors of off limit rectangles
		// rectangle #1
		//offlimits is the size of cfg
		vertex rect1Min = minValue(vertices, offlimits[j].point1Index, cfg[j].x, cfg[j].y);
		vertex rect1Max = maxValue(vertices, offlimits[j].point1Index, cfg[j].x, cfg[j].y);

		// printf("Clearance rectangle %d: Min X: %f Y: %f Max X: %f Y: %f\n", i, rect1Min.x, rect1Min.y, rect1Max.x, rect1Max.y);
		values -= calculateIntersectionArea(rect1Min, rect1Max, complementRectangle1[0], complementRectangle1[1]);
		values -= calculateIntersectionArea(rect1Min, rect1Max, complementRectangle2[0], complementRectangle2[1]);
		values -= calculateIntersectionArea(rect1Min, rect1Max, complementRectangle3[0], complementRectangle3[1]);
		values -= calculateIntersectionArea(rect1Min, rect1Max, complementRectangle4[0], complementRectangle4[1]);
	}

	group.sync();
	reduce<tile_sz>(group, values);
	//printf("Clearance costs error: %f\n", error);
	return values;
}

template<int tile_sz>
__device__ float OffLimitsCosts(cg::thread_block_tile<tile_sz> group, Surface *srf, positionAndRotation *cfg, vertex *vertices, rectangle *offlimits) {
	int tid = group.thread_rank();
	int step = group.size();
	float values;
	values = 0.0f; //Since it's size blockDim, we can have each of them treat it as the starting value
	for (int i = tid; i < srf->nObjs; i += step) {
		vertex rect1Min = minValue(vertices, offlimits[i].point1Index, cfg[i].x, cfg[i].y);
		vertex rect1Max = maxValue(vertices, offlimits[i].point1Index, cfg[i].x, cfg[i].y);
		for (int j = i + 1; j < srf->nObjs; j++) {
			// Determine max and min vectors of clearance rectangles
			// rectangle #1
			//printf("Clearance\n");
			//printf("Translation: X: %f Y: %f\n", cfg[i].x, cfg[i].y);
			// printf("Clearance rectangle %d: Min X: %f Y: %f Max X: %f Y: %f\n", i, rect1Min.x, rect1Min.y, rect1Max.x, rect1Max.y);
			// rectangle #2
			//printf("Off limits\n");
			//printf("Translation: X: %f Y: %f\n", cfg[j].x, cfg[j].y);
			//offlimits is the size of cfg
			vertex rect2Min = minValue(vertices, offlimits[j].point1Index, cfg[j].x, cfg[j].y);
			vertex rect2Max = maxValue(vertices, offlimits[j].point1Index, cfg[j].x, cfg[j].y);

			// printf("Clearance rectangle %d: Min X: %f Y: %f Max X: %f Y: %f\n", i, rect2Min.x, rect2Min.y, rect2Max.x, rect2Max.y);

			float area = calculateIntersectionArea(rect1Min, rect1Max, rect2Min, rect2Max);
			//printf("Area intersection rectangle %d and %d: %f\n", i, j, area);
			values -= area;
		}

	}

	group.sync();
	reduce<tile_sz>(group, values);
	//printf("Clearance costs error: %f\n", error);
	return values;
}

template<int tile_sz>
__device__ void Costs(cg::thread_block_tile<tile_sz> group, Surface *srf, resultCosts* costs, positionAndRotation* cfg, relationshipStruct *rs, vertex *vertices, rectangle *clearances, rectangle *offlimits, vertex *surfaceRectangle)
{
	int gid = group.thread_rank();
	//float pairWiseCosts = 0;
	float pairWiseCosts =  PairWiseTotalCosts<tile_sz>(group, srf, cfg, rs);
	pairWiseCosts *= srf->WeightPairWise;

	// printf("Pair wise costs with weight %f\n", pairWiseCosts);

	//float visualBalanceCosts = 0;
	float visualBalanceCosts = srf->WeightVisualBalance * VisualBalanceCosts<tile_sz>(group, srf, cfg);
	
	// printf("Visual balance costs with weight %f\n", visualBalanceCosts);

	//float focalPointCosts = 0;
	float focalPointCosts = srf->WeightFocalPoint * FocalPointCosts<tile_sz>(group, srf, cfg);
	
	// printf("Focal point costs with weight %f\n", focalPointCosts);

	//float symmertryCosts = 0;
	float symmertryCosts = srf->WeightSymmetry * SymmetryCosts<tile_sz>(group, srf, cfg);
	
	// printf("Symmertry costs with weight %f\n", symmertryCosts);

	//float offlimitsCosts = 0;
	float offlimitsCosts = srf->WeightOffLimits * OffLimitsCosts<tile_sz>(group, srf, cfg, vertices, offlimits);
	// printf("OffLimits costs with weight %f\n", offlimitsCosts);
	

	//float clearanceCosts = 0;
	float clearanceCosts = srf->WeightClearance * ClearanceCosts<tile_sz>(group, srf, cfg, vertices, clearances, offlimits);
	// printf("Clearance costs with weight %f\n", clearanceCosts);
	

	//float surfaceAreaCosts = 0;
	float surfaceAreaCosts = srf->WeightSurfaceArea * SurfaceAreaCosts<tile_sz>(group, srf, cfg, vertices, clearances, offlimits, surfaceRectangle);
	// printf("Surface area costs with weight %f\n", surfaceAreaCosts);
	
	float totalCosts = pairWiseCosts + visualBalanceCosts + focalPointCosts + symmertryCosts + clearanceCosts + surfaceAreaCosts;
	if (gid == 0) {
		costs->PairWiseCosts = pairWiseCosts;
		costs->VisualBalanceCosts = visualBalanceCosts;
		costs->FocalPointCosts = focalPointCosts;
		costs->SymmetryCosts = symmertryCosts;
		costs->OffLimitsCosts = offlimitsCosts;
		costs->ClearanceCosts = clearanceCosts;
		costs->SurfaceAreaCosts = surfaceAreaCosts;
		costs->totalCosts = totalCosts;
	}
	group.sync();
	
	// printf("Total costs %f\n", totalCosts);
	
}

__device__ void CopyCosts(resultCosts* copyFrom, resultCosts* copyTo) 
{
	copyTo->PairWiseCosts = copyFrom->PairWiseCosts;
	copyTo->VisualBalanceCosts = copyFrom->VisualBalanceCosts;
	copyTo->FocalPointCosts = copyFrom->FocalPointCosts;
	copyTo->SymmetryCosts = copyFrom->SymmetryCosts;
	//printf("Copying Clearance costs with weight %f\n", copyFrom->ClearanceCosts);
	copyTo->ClearanceCosts = copyFrom->ClearanceCosts;
	copyTo->OffLimitsCosts = copyFrom->OffLimitsCosts;
	//printf("Copying Surface area costs with weight %f\n", copyFrom->SurfaceAreaCosts);
	copyTo->SurfaceAreaCosts = copyFrom->SurfaceAreaCosts;
	copyTo->totalCosts = copyFrom->totalCosts;
}

//The next two device helper functions generate random numbers
__inline__ __device__ int generateRandomIntInRange(hiprandState *rngStates, unsigned int tid, int max, int min)
{
	hiprandState localState = rngStates[tid];
	float p_rand = hiprand_uniform(&localState);
	rngStates[tid] = localState;
	p_rand *= (max - min + 0.999999);
	p_rand += min;
	return (int)truncf(p_rand);
}

__inline__ __device__ float generateRandomFloatInRange(hiprandState *rngStates, unsigned int tid, int max, int min)
{
	hiprandState localState = rngStates[tid];
	float p_rand = hiprand_uniform(&localState);
	rngStates[tid] = localState;
	p_rand *= (max - min + 0.999999);
	p_rand += min;
	return p_rand; //The only difference between float and int is that we do not trucate the float in this one
}

template<int tile_sz>
__device__ void propose(cg::thread_block_tile<tile_sz> group, Surface *srf, positionAndRotation *cfg, vertex * surfaceRectangle, hiprandState *rngStates, unsigned int tid)
{
	int gid = group.thread_rank();
	/*for (int j = 0; j < srf->nObjs; j++)
	{
		printf("Star values inside proposition jndex %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", j, cfgStar[j].x, cfgStar[j].y, cfgStar[j].z, cfgStar[j].rotX, cfgStar[j].rotY, cfgStar[j].rotZ);
	}*/
	int p = generateRandomIntInRange(rngStates, tid, 2, 0);

	//Get everyone on the same page
	p = group.shfl(p, 0); //broadcast out to p
	//group.sync(); shlf_sync is broadcast and so should sync
	// Determine width and length of surface rectangle
	vertex srfRect1Min = minValue(surfaceRectangle, 0, 0, 0);
	vertex srfRect1Max = maxValue(surfaceRectangle, 0, 0, 0);
	float width = srfRect1Max.x - srfRect1Min.x;
	float height = srfRect1Max.y - srfRect1Min.y;
	// Dividing the width by 2 makes sure that it stays withing a 95% percentile range that is usable, dividing it by 4 makes sure that it stretches the half of the length/width or lower (and that inside a 95% interval).
	float stdXAxis = width / 16;
	float stdYAxis = height / 16;

	// printf("Selected mode: %d\n", p);
	// Translate location using normal distribution
	if (p == 0)
	{
		//bool found = false;
		int obj = -1;
		// Take 100 tries to find a random nonfrozen object
		//for (int i = 0; i < 100 && !found; i++) {
		obj = generateRandomIntInRange(rngStates, tid, srf->nObjs - 1, 0);
		if (!group.any(!cfg[obj].frozen)) {
			return;
		}
		int mask = group.ballot(!cfg[obj].frozen);
		int leader = __ffs(mask);
		obj = group.shfl(obj, leader);


		//printf("Selected object #: %d\n", obj);
		if (gid == 0) {
			float dx = hiprand_normal(&rngStates[tid]);
			dx = dx * stdXAxis;
			//printf("dx: %f\n", dx);
			float dy = hiprand_normal(&rngStates[tid]);
			dy = dy * stdYAxis;
			// printf("Before translation, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj, cfgStar[obj].x, cfgStar[obj].y, cfgStar[obj].z, cfgStar[obj].rotX, cfgStar[obj].rotY, cfgStar[obj].rotZ);

			// When object exceeds surfacearea, snap it back.
			if (cfg[obj].x + dx > srfRect1Max.x) {
				cfg[obj].x = srfRect1Max.x;
			}
			else if (cfg[obj].x + dx < srfRect1Min.x) {
				cfg[obj].x = srfRect1Min.x;
			}
			else {
				cfg[obj].x += dx;
			}
			if (cfg[obj].y + dy > srfRect1Max.y) {
				cfg[obj].y = srfRect1Max.y;
			}
			else if (cfg[obj].y + dy < srfRect1Min.y) {
				cfg[obj].y = srfRect1Min.y;
			}
			else {
				cfg[obj].y += dy;
			}
		}
		// printf("After rotation, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj, cfgStar[obj].x, cfgStar[obj].y, cfgStar[obj].z, cfgStar[obj].rotX, cfgStar[obj].rotY, cfgStar[obj].rotZ);
	}
	// Translate rotation using normal distribution
	else if (p == 1)
	{
		int obj = -1;
		// Take 100 tries to find a random nonfrozen object
		//for (int i = 0; i < 100 && !found; i++) {
		obj = generateRandomIntInRange(rngStates, tid, srf->nObjs - 1, 0);
		if (!group.any(!cfg[obj].frozen)) {
			return;
		}
		int mask = group.ballot(!cfg[obj].frozen);
		int leader = __ffs(mask);
		obj = group.shfl(obj, leader);


		if (gid == 0) {
			// printf("Selected object #: %d\n", obj);
			// printf("Before rotation, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj, cfgStar[obj].x, cfgStar[obj].y, cfgStar[obj].z, cfgStar[obj].rotX, cfgStar[obj].rotY, cfgStar[obj].rotZ);
			float dRot = hiprand_normal(&rngStates[tid]);
			dRot = dRot * S_SIGMA_T;
			// printf("dRot: %f\n", dRot);
			// printf("before rotation: %f\n", cfgStar[obj].rotY);
			cfg[obj].rotY += dRot;
			// printf("After rotation: %f\n", cfgStar[obj].rotY);

			if (cfg[obj].rotY < 0)
				cfg[obj].rotY += 2 * PI;
			else if (cfg[obj].rotY > 2 * PI)
				cfg[obj].rotY -= 2 * PI;
		}
		// printf("After rotation, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj, cfgStar[obj].x, cfgStar[obj].y, cfgStar[obj].z, cfgStar[obj].rotX, cfgStar[obj].rotY, cfgStar[obj].rotZ);
	}
	// Swap two objects for both location and rotation
	else
	{
		if (srf->nObjs < 2) {
			return;
		}
		// This can result in the same object, chance becomes increasingly smaller given more objects
		int obj1 = -1;
		int obj2 = -1;
		obj1 = generateRandomIntInRange(rngStates, tid, srf->nObjs - 1, 0);
		if (!group.any(!cfg[obj1].frozen)) {
			return;
		}
		int mask = group.ballot(!cfg[obj1].frozen);
		int leader = __ffs(mask);
		obj1 = group.shfl(obj1, leader);

		obj2 = generateRandomIntInRange(rngStates, tid, srf->nObjs - 1, 0);
		if (!group.any(!cfg[obj2].frozen)) {
			return;
		}
		mask = group.ballot(!cfg[obj2].frozen);
		leader = __ffs(mask);
		obj2 = group.shfl(obj2, leader);

		if (obj1 == obj2) {
			return; //No point at this step
		}
		// printf("First selected object #: %d\n", obj1);
		// printf("Second selected object #: %d\n", obj2);

		// printf("Values, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj1, cfgStar[obj1].x, cfgStar[obj1].y, cfgStar[obj1].z, cfgStar[obj1].rotX, cfgStar[obj1].rotY, cfgStar[obj1].rotZ);
		// printf("Values of, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj2, cfgStar[obj2].x, cfgStar[obj2].y, cfgStar[obj2].z, cfgStar[obj2].rotX, cfgStar[obj2].rotY, cfgStar[obj2].rotZ);
		if (gid == 0) {
			// Temporarily store cfgStar[obj1] values
			float x = cfg[obj1].x;
			float y = cfg[obj1].y;
			float z = cfg[obj1].z;
			float rotX = cfg[obj1].rotX;
			float rotY = cfg[obj1].rotY;
			float rotZ = cfg[obj1].rotZ;
			// printf("After copy obj1 to temp, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj1, cfgStar[obj1].x, cfgStar[obj1].y, cfgStar[obj1].z, cfgStar[obj1].rotX, cfgStar[obj1].rotY, cfgStar[obj1].rotZ);
			// printf("After copy obj1 to temp, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj2, cfgStar[obj2].x, cfgStar[obj2].y, cfgStar[obj2].z, cfgStar[obj2].rotX, cfgStar[obj2].rotY, cfgStar[obj2].rotZ);

			// Move values of obj2 to obj1
			cfg[obj1].x = cfg[obj2].x;
			cfg[obj1].y = cfg[obj2].y;
			cfg[obj1].z = cfg[obj2].z;
			cfg[obj1].rotX = cfg[obj2].rotX;
			cfg[obj1].rotY = cfg[obj2].rotY;
			cfg[obj1].rotZ = cfg[obj2].rotZ;
			// printf("After copy obj2 into obj1, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj1, cfgStar[obj1].x, cfgStar[obj1].y, cfgStar[obj1].z, cfgStar[obj1].rotX, cfgStar[obj1].rotY, cfgStar[obj1].rotZ);
			// printf("After copy obj2 into obj1, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj2, cfgStar[obj2].x, cfgStar[obj2].y, cfgStar[obj2].z, cfgStar[obj2].rotX, cfgStar[obj2].rotY, cfgStar[obj2].rotZ);

			// Move stored values of obj1 to obj2
			cfg[obj2].x = x;
			cfg[obj2].y = y;
			cfg[obj2].z = z;
			cfg[obj2].rotX = rotX;
			cfg[obj2].rotY = rotY;
			cfg[obj2].rotZ = rotZ;
		}
		// printf("After copy temp into obj2, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj1, cfgStar[obj1].x, cfgStar[obj1].y, cfgStar[obj1].z, cfgStar[obj1].rotX, cfgStar[obj1].rotY, cfgStar[obj1].rotZ);
		// printf("After copy temp into obj2, obj %d. X, Y, Z: %f, %f, %f rotation: %f, %f, %f\n", obj2, cfgStar[obj2].x, cfgStar[obj2].y, cfgStar[obj2].z, cfgStar[obj2].rotX, cfgStar[obj2].rotY, cfgStar[obj2].rotZ);
	}
}

__device__ bool Accept(double costStar, double costCur, hiprandState *rngStates, unsigned int tid,float beta)
{
	//printf("(costStar - costCur):  %f\n", (costStar - costCur));
	//printf("(float) exp(-BETA * (costStar - costCur)): %f\n", (float)exp(-BETA * (costStar - costCur)));
	float randomNumber = hiprand_uniform(&rngStates[tid]);
	//printf("Random number: %f\n", randomNumber);
	return  randomNumber < fminf(1.0f, (float) exp(beta * (costStar - costCur)));
}

template<int tile_sz>
__device__ void Copy(cg::thread_block_tile<tile_sz> group, positionAndRotation* cfg1, positionAndRotation* cfg2, Surface* srf)
{
	int tid = group.thread_rank();
	int step = group.size();
	for (unsigned int i = tid; i < srf->nObjs; i += step)
	{
		cfg1[i].x = cfg2[i].x;
		cfg1[i].y = cfg2[i].y;
		cfg1[i].z = cfg2[i].z;
		cfg1[i].rotX = cfg2[i].rotX;
		cfg1[i].rotY = cfg2[i].rotY;
		cfg1[i].rotZ = cfg2[i].rotZ;
		cfg1[i].frozen = cfg2[i].frozen;
		cfg1[i].length = cfg2[i].length;
		cfg1[i].width = cfg2[i].width;
	}
	group.sync();
}

template<int tile_sz>
__device__ void groupKernel(cg::thread_block_tile<tile_sz> group,
	positionAndRotation* cfgBest,
	resultCosts* bestCosts,
	positionAndRotation* cfgStar,
	resultCosts* starCosts,
	relationshipStruct *rs,
	rectangle *clearances, rectangle *offlimits,
	vertex *vertices, vertex *surfaceRectangle, Surface *srf,
	int iterations, hiprandState *rngStates)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int gtid = group.thread_rank();//The thread id in the working group
	int step = group.size();//The working group block size
							// Read out starting configuration from resultArray
							// Copy best config (now set to input config) to result of this block

	bool accept;
	float beta = generateRandomFloatInRange(rngStates, tid, 3, 0);
	beta = group.shfl(beta, 0);//shf calls shfl_sync, which as a broadcast should sync
	Copy<tile_sz>(group, cfgStar, cfgBest, srf);

	Costs<tile_sz>(group, srf, bestCosts, cfgBest, rs, vertices, clearances, offlimits, surfaceRectangle); //possible race condition here
	CopyCosts(bestCosts, starCosts);
	//printf("Threadblock: %d, Best costs before: %f\n", blockIdx.x, bestCosts->totalCosts);

	for (int i = 0; i < iterations; i++)
	{
		
		propose<tile_sz>(group, srf, cfgStar, surfaceRectangle, rngStates, tid);
		group.sync();
		Costs<tile_sz>(group, srf, starCosts, cfgStar, rs, vertices, clearances, offlimits, surfaceRectangle);
		if (gtid == 0) {
			accept = Accept(starCosts->totalCosts, bestCosts->totalCosts, rngStates, tid,beta);

		}
		accept = group.shfl(accept, 0);
		if (accept)
		{
			// Possible different approach: Set pointer of current to star, free up memory used by current? reinitialize star?
			//printf("Star accepted as new current.\n");
			// Copy star into current
			Copy<tile_sz>(group, cfgBest, cfgStar, srf);
			CopyCosts(starCosts, bestCosts);
		}
		else { //Reject it
			Copy<tile_sz>(group, cfgStar, cfgBest, srf);
			CopyCosts(bestCosts, starCosts);
		}

		// Check whether we continue with current or we continue with star
	}

	group.sync();
}

//Helper function to copy the information from global memory into a shared array
template<int tile_sz>
__device__ void copyToSharedMemory(cg::thread_block_tile<tile_sz> group,
	point *p,
	Surface *srf,
	positionAndRotation* configuration) {
	int gid = group.thread_rank();
	for (unsigned int i = gid; i < srf->nObjs; i += WARP_SIZE)
	{
		// BlockId counts from 0, so to properly multiply
		int index = blockIdx.x * srf->nObjs + i;
		configuration[i].x = p[index].x;
		configuration[i].y = p[index].y;
		configuration[i].z = p[index].z;
		configuration[i].rotX = p[index].rotX;
		configuration[i].rotY = p[index].rotY;
		configuration[i].rotZ = p[index].rotZ;
		configuration[i].frozen = p[index].frozen;
		configuration[i].length = p[index].length;
		configuration[i].width = p[index].width;
	}

}
//Helper function to copy the information from shared to global.
__device__ void copyToGlobalMemory(
	point *p,
	Surface *srf,
	resultCosts* resultCostsArray,
	positionAndRotation* configuration,
	resultCosts* costs,
	int lowest_cost) {
	//Copy current config back into the global memory
	// Copy best config (now set to input config) to result of this block
	for (unsigned int i = threadIdx.x; i < srf->nObjs; i += blockDim.x)
	{
		// BlockId counts from 0, so to properly multiply
		int index = blockIdx.x * srf->nObjs + i;
		p[index].x = configuration[lowest_cost * srf->nObjs + i].x;
		p[index].y = configuration[lowest_cost * srf->nObjs + i].y;
		p[index].z = configuration[lowest_cost * srf->nObjs + i].z;
		p[index].rotX = configuration[lowest_cost * srf->nObjs + i].rotX;
		p[index].rotY = configuration[lowest_cost * srf->nObjs + i].rotY;
		p[index].rotZ = configuration[lowest_cost * srf->nObjs + i].rotZ;
		p[index].frozen = configuration[lowest_cost * srf->nObjs + i].frozen;
		p[index].length = configuration[lowest_cost * srf->nObjs + i].length;
		p[index].width = configuration[lowest_cost * srf->nObjs + i].width;
	}
	//printf("Threadblock: %d, Result costs before: %f\n", blockIdx.x, bestCosts->totalCosts);
	resultCostsArray[blockIdx.x].totalCosts = costs[lowest_cost].totalCosts;
	resultCostsArray[blockIdx.x].PairWiseCosts = costs[lowest_cost].PairWiseCosts;
	resultCostsArray[blockIdx.x].VisualBalanceCosts = costs[lowest_cost].VisualBalanceCosts;
	resultCostsArray[blockIdx.x].FocalPointCosts = costs[lowest_cost].FocalPointCosts;
	resultCostsArray[blockIdx.x].SymmetryCosts = costs[lowest_cost].SymmetryCosts;
	//printf("Best surface area costs: %f\n", bestCosts->SurfaceAreaCosts);
	resultCostsArray[blockIdx.x].SurfaceAreaCosts = costs[lowest_cost].SurfaceAreaCosts;
	//printf("Best clearance costs: %f\n", bestCosts->ClearanceCosts);
	resultCostsArray[blockIdx.x].ClearanceCosts = costs[lowest_cost].ClearanceCosts;
	resultCostsArray[blockIdx.x].OffLimitsCosts = costs[lowest_cost].OffLimitsCosts;

}

//This function figures out the lowest cost of our search
//It can be written as a reduction problem, and definitely should
__device__ int lowestIndex(resultCosts* best_costs, int active_warps) {
	int best_cost = 0;
	for (int i = 0; i < active_warps; i++) {
		if (best_costs[i].totalCosts < best_costs[best_cost].totalCosts) {
			best_cost = i;
		}
	}
	return best_cost;
}
// result is a [,] array with 1 dimension equal to the amount of blocks used and the other dimension equal to the amount of objects
// rs is an array with the length equal to the amount of relationships
// cfg is an array with the length equal to the amount of objects
// Surface is a basic struct

__global__ void Kernel(resultCosts* resultCostsArray,
	point *p, relationshipStruct *rs,
	rectangle *clearances, rectangle *offlimits,
	vertex *vertices, vertex *surfaceRectangle, Surface *srf,
	gpuConfig *gpuCfg, hiprandState *rngStates) {

	extern __shared__ int all_shared_memory[];
	int jumper = blockDim.x / WARP_SIZE;
	positionAndRotation* configurations = (positionAndRotation*)&all_shared_memory;
	resultCosts* costs = (resultCosts*)&configurations[2 * jumper * srf->nObjs]; 
	__syncthreads();
   //create the working groups
	int rank = threadIdx.x / WARP_SIZE;
	positionAndRotation* best_conf = &configurations[rank * srf->nObjs];
	positionAndRotation* star_conf = &configurations[srf->nObjs * blockDim.x / WARP_SIZE + rank];
	resultCosts* best_cost = &costs[rank];
	resultCosts* star_cost = &costs[jumper + rank];
	auto tile_warp = cg::tiled_partition<WARP_SIZE>(cg::this_thread_block()); //Broken up by our warp size, which is our static shared memory size!

	//This is the actual work done
	copyToSharedMemory<WARP_SIZE>(tile_warp, p, srf, best_conf);
	groupKernel<WARP_SIZE>(tile_warp,best_conf,best_cost,star_conf,star_cost, rs, clearances, offlimits, vertices, surfaceRectangle, srf, gpuCfg->iterations, rngStates);
	__syncthreads();
	int lowest_cost = lowestIndex(best_cost, jumper);
	copyToGlobalMemory(p, srf, resultCostsArray, best_conf, costs, lowest_cost);
	__syncthreads();

}


extern "C" __declspec(dllexport) result* KernelWrapper(relationshipStruct *rss, point *previouscfgs, rectangle *clearances, rectangle *offlimits, vertex *vertices, vertex *surfaceRectangle, Surface *srf, gpuConfig *gpuCfg)
{
	// Create pointer for on gpu
	// Determine memory size of object to transfer
	// Malloc on GPU size
	// Cpy memory from cpu to gpu
	relationshipStruct *gpuRS;
	int rsSize = sizeof(relationshipStruct) * srf->nRelationships;
	checkCudaErrors(hipMalloc(&gpuRS, rsSize));
	checkCudaErrors(hipMemcpy(gpuRS, rss, rsSize, hipMemcpyHostToDevice));

	// Input
	rectangle *gpuClearances;
	int clearancesSize = sizeof(rectangle) * srf->nClearances;
	checkCudaErrors(hipMalloc(&gpuClearances, clearancesSize));
	checkCudaErrors(hipMemcpy(gpuClearances, clearances, clearancesSize, hipMemcpyHostToDevice));

	rectangle *gpuOfflimits;
	int offlimitsSize = sizeof(rectangle) * srf->nObjs;
	checkCudaErrors(hipMalloc(&gpuOfflimits, offlimitsSize));
	checkCudaErrors(hipMemcpy(gpuOfflimits, offlimits, offlimitsSize, hipMemcpyHostToDevice));

	vertex *gpuVertices;
	int verticesSize = sizeof(vertex) * (srf->nClearances * 4 + srf->nObjs * 4);
	checkCudaErrors(hipMalloc(&gpuVertices, verticesSize));
	checkCudaErrors(hipMemcpy(gpuVertices, vertices, verticesSize, hipMemcpyHostToDevice));

	vertex *gpuSurfaceRectangle;
	int surfaceRectangleSize = sizeof(vertex) * 4;
	checkCudaErrors(hipMalloc(&gpuSurfaceRectangle, surfaceRectangleSize));
	checkCudaErrors(hipMemcpy(gpuSurfaceRectangle, surfaceRectangle, surfaceRectangleSize, hipMemcpyHostToDevice));

	Surface *gpuSRF;
	int srfSize = sizeof(Surface);
	checkCudaErrors(hipMalloc(&gpuSRF, srfSize));
	checkCudaErrors(hipMemcpy(gpuSRF, srf, srfSize, hipMemcpyHostToDevice));

	gpuConfig *gpuGpuConfig;
	int gpuCFGSize = sizeof(gpuConfig);
	checkCudaErrors(hipMalloc(&gpuGpuConfig, gpuCFGSize));
	checkCudaErrors(hipMemcpy(gpuGpuConfig, gpuCfg, gpuCFGSize, hipMemcpyHostToDevice));

	// Output
	point *gpuPointArray;
	int pointArraySize = srf->nObjs * sizeof(point) * gpuCfg->gridxDim;
	point *outPointArray = (point *) malloc(pointArraySize);
	checkCudaErrors(hipMalloc((void**)&gpuPointArray, pointArraySize));
	checkCudaErrors(hipMemcpy(gpuPointArray, previouscfgs, pointArraySize, hipMemcpyHostToDevice));

	resultCosts *gpuResultCosts;
	int resultCostsSize = sizeof(resultCosts) * gpuCfg->gridxDim;
	resultCosts *outResultCosts = (resultCosts *)malloc(resultCostsSize);
	checkCudaErrors(hipMalloc((void**)&gpuResultCosts, resultCostsSize));

	// hipMemcpy(gpuPointArray, result, pointArraySize, hipMemcpyHostToDevice);
	//Size of the shared array that holds the configuration data
	
	// Setup GPU random generator
	hiprandState *d_rngStates = 0;
	checkCudaErrors(hipMalloc((void **)&d_rngStates, gpuCfg->gridxDim * gpuCfg->blockxDim * sizeof(hiprandState)));

	// Initialise random number generator
	initRNG <<<gpuCfg->gridxDim, gpuCfg->blockxDim>> > (d_rngStates, time(NULL));

	// Commented for possible later usage
	// dim3 dimGrid(gpuCfg->gridxDim, gpuCfg->gridyDim);
	// dim3 dimBlock(gpuCfg->blockxDim, gpuCfg->blockyDim, gpuCfg->blockzDim);
	
	// Block 1 dimensional, amount of threads available, configurable
	// Grid 1 dimension, amount of suggestions to be made.
	//we make the dynamic memory 3 times because we have at least 3 arrays that use it in one function
	int share_size = gpuCfg->blockxDim / WARP_SIZE * 2 * srf->nObjs * sizeof(positionAndRotation) + gpuCfg->blockxDim / WARP_SIZE * 2 * sizeof(resultCosts);
	Kernel <<<gpuCfg->gridxDim, gpuCfg->blockxDim,share_size>>>(gpuResultCosts, gpuPointArray, gpuRS, gpuClearances, gpuOfflimits, gpuVertices, gpuSurfaceRectangle, gpuSRF, gpuGpuConfig, d_rngStates);
	checkCudaErrors(hipDeviceSynchronize());
	if (hipSuccess != hipGetLastError()) {
		fprintf(stderr, "cudaSafeCall() failed : %s\n",
			hipGetErrorString(hipGetLastError()));
	}

	// copy back results from gpu to cpu
	checkCudaErrors(hipMemcpy(outPointArray, gpuPointArray, pointArraySize, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(outResultCosts, gpuResultCosts, resultCostsSize, hipMemcpyDeviceToHost));

	// Free all allocated GPU memory
	hipFree(gpuRS);
	hipFree(gpuClearances);
	hipFree(gpuOfflimits);
	hipFree(gpuVertices);
	hipFree(gpuSurfaceRectangle);
	hipFree(gpuSRF);
	hipFree(gpuGpuConfig);
	hipFree(gpuPointArray);
	hipFree(gpuResultCosts);

	// Construct return result
	result *resultPointer = (result*)malloc(sizeof(result) * gpuCfg->gridxDim);
	for (int i = 0; i < gpuCfg->gridxDim; i++)
	{
		resultPointer[i].costs.FocalPointCosts = outResultCosts[i].FocalPointCosts;
		resultPointer[i].costs.PairWiseCosts = outResultCosts[i].PairWiseCosts;
		resultPointer[i].costs.SymmetryCosts = outResultCosts[i].SymmetryCosts;
		resultPointer[i].costs.totalCosts = outResultCosts[i].totalCosts;
		resultPointer[i].costs.VisualBalanceCosts = outResultCosts[i].VisualBalanceCosts;
		resultPointer[i].costs.ClearanceCosts = outResultCosts[i].ClearanceCosts;
		resultPointer[i].costs.OffLimitsCosts = outResultCosts[i].OffLimitsCosts;
		resultPointer[i].costs.SurfaceAreaCosts = outResultCosts[i].SurfaceAreaCosts;
		resultPointer[i].points = &(outPointArray[i * srf->nObjs]);
	}
	return resultPointer;
}

void basicCudaDeviceInformation(int argc, char **argv) {
	int devID;
	hipDeviceProp_t props;

	// This will pick the best possible CUDA capable device
	devID = findCudaDevice(argc, (const char **)argv);

	//Get GPU information
	checkCudaErrors(hipGetDevice(&devID));
	checkCudaErrors(hipGetDeviceProperties(&props, devID));
	printf("Device %d: \"%s\" with Compute %d.%d capability\n",
		devID, props.name, props.major, props.minor);

	printf("printf() is called. Output:\n\n");
}


int main(int argc, char **argv)
{
	basicCudaDeviceInformation(argc, argv);

	const int N = 10;
	const int NRel = 1;
	const int NClearances = 30;
	Surface srf;
	srf.nObjs = N;
	srf.nRelationships = NRel;
	srf.nClearances = NClearances;
	srf.WeightFocalPoint = 1.0f;
	srf.WeightPairWise = 1.0f;
	srf.WeightVisualBalance = 1.0f;
	srf.WeightSymmetry = 1.0f;
	srf.WeightClearance = 1.0f;
	srf.WeightSurfaceArea = 1.0f;
	srf.WeightOffLimits = 1.0f;
	srf.centroidX = 0.0;
	srf.centroidY = 0.0;
	srf.focalX = 5.0;
	srf.focalY = 5.0;
	srf.focalRot = 0.0;

	const int dimensions = 8;

	gpuConfig gpuCfg;

	gpuCfg.gridxDim = dimensions;
	gpuCfg.gridyDim = 0;
	gpuCfg.blockxDim = 4*WARP_SIZE;
	gpuCfg.blockyDim = 0;
	gpuCfg.blockzDim = 0;
	gpuCfg.iterations = 1000;//a 10th of what they claimed in the paper

	vertex surfaceRectangle[4];
	surfaceRectangle[0].x = 10;
	surfaceRectangle[0].y = 10;
	surfaceRectangle[0].z = 0;

	surfaceRectangle[1].x = 10;
	surfaceRectangle[1].y = 0;
	surfaceRectangle[1].z = 0;

	surfaceRectangle[2].x = 0;
	surfaceRectangle[2].y = 0;
	surfaceRectangle[2].z = 0;

	surfaceRectangle[3].x = 0;
	surfaceRectangle[3].y = 10;
	surfaceRectangle[3].z = 0;

	const int vertices = (N + NClearances) * 4;
	vertex vtx[vertices];
	for (int i = 0; i < (N); i++) {
		vtx[i*16+0].x = -1.8853001594543457;
		vtx[i*16 + 0].y = 1.1240049600601196;
		vtx[i*16 +0].z = 0;

		vtx[i*16 + 1].x = -0.88530009984970093;
		vtx[i*16 + 1].y = 1.1240049600601196;
		vtx[i*16 + 1].z = 0;

		vtx[i*16 + 2].x = -0.88530009984970093;
		vtx[i*16 + 2].y = -1.1240470409393311;
		vtx[i*16 + 2].z = 0;

		vtx[i*16 + 3].x = -1.8853001594543457;
		vtx[i*16 + 3].y = -1.1240470409393311;
		vtx[i*16 + 3].z = 0;

		vtx[i*16 + 4].x = 0.88240820169448853;
		vtx[i*16 + 4].y = 1.1240049600601196;
		vtx[i*16 + 4].z = 0;

		vtx[i*16 + 5].x = 1.8824081420898437;
		vtx[i*16 + 5].y = 1.1240049600601196;
		vtx[i*16 + 5].z = 0;

		vtx[i*16 + 6].x = 1.8824081420898437;
		vtx[i*16 + 6].y = -1.1240470409393311;
		vtx[i*16 + 6].z = 0;

		vtx[i*16 + 7].x = 0.88240820169448853;
		vtx[i*16 + 7].y = -1.1240470409393311;
		vtx[i*16 + 7].z = 0;

		vtx[i*16 + 8].x = -0.88530009984970093;
		vtx[i*16 + 8].y = 2.12400484085083;
		vtx[i*16 + 8].z = 0;

		vtx[i*16 + 9].x = 0.88240820169448853;
		vtx[i*16 + 9].y = 2.12400484085083;
		vtx[i*16 + 9].z = 0;

		vtx[i*16 + 10].x = 0.88240820169448853;
		vtx[i*16 + 10].y = 1.1240049600601196;
		vtx[i*16 + 10].z = 0;

		vtx[i*16 + 11].x = -0.88530009984970093;
		vtx[i*16 + 11].y = 1.1240049600601196;
		vtx[i*16 + 11].z = 0;

		vtx[i*16 + 12].x = -7.3193349838256836;
		vtx[i*16 + 12].y = -0.99961233139038086;
		vtx[i*16 + 12].z = 1.2984378337860107;

		vtx[i*16 + 13].x = -5.5516266822814941;
		vtx[i*16 + 13].y = -0.99961233139038086;
		vtx[i*16 + 13].z = 1.2984378337860107;

		vtx[i*16 + 14].x = -5.5516266822814941;
		vtx[i*16 + 14].y = -3.2476644515991211;
		vtx[i*16 + 14].z = 1.2984378337860107;

		vtx[i*16 + 15].x = -7.3193349838256836;
		vtx[i*16 + 15].y = -3.2476644515991211;
		vtx[i*16 + 15].z = 1.2984378337860107;
	}

	rectangle clearances[NClearances];
	rectangle offlimits[N];
	for (int i = 0; i < N; i++) {
		clearances[i*3].point1Index = i*15 + 0;
		clearances[i * 3].point2Index = i * 15 + 1;
		clearances[i * 3].point3Index = i * 15 + 2;
		clearances[i * 3].point4Index = i * 15 + 3;
		clearances[i * 3].SourceIndex = i;

		clearances[i * 3  + 1].point1Index = i * 15 + 4;
		clearances[i * 3 + 1].point2Index = i * 15 + 5;
		clearances[i * 3 + 1].point3Index = i * 15 + 6;
		clearances[i * 3 + 1].point4Index = i * 15 + 7;
		clearances[i * 3 + 1].SourceIndex = i;

		clearances[i * 3 + 2].point1Index = i * 15 + 8;
		clearances[i * 3 + 2].point2Index = i * 15 + 9;
		clearances[i * 3 + 2].point3Index = i * 15 + 10;
		clearances[i * 3 + 2].point4Index = i * 15 + 11;
		clearances[i * 3 + 2].SourceIndex = i;

		offlimits[i].point1Index = i * 15 + 12;
		offlimits[i].point2Index = i * 15 + 13;
		offlimits[i].point3Index = i * 15 + 14;
		offlimits[i].point4Index = i * 15 + 15;
		offlimits[i].SourceIndex = 0;
	}

	point cfg[N*dimensions];
	for (int i = 0; i < dimensions; i++) {
		for (unsigned int j = 0; j < N; j++)
		{
			// BlockId counts from 0, so to properly multiply
			int index = i * N + j;

			cfg[index].x = -6.4340348243713379;
			cfg[index].y = -2.12361741065979;
			cfg[index].z = 0.0;
			cfg[index].rotX = 0.0;
			cfg[index].rotY = 5.5179219245910645;
			cfg[index].rotZ = 0.0;
			cfg[index].frozen = false;
			cfg[index].length = 1.7677083015441895;
			cfg[index].width = 2.2480521202087402;
		}
	}

	// Create relationship
	relationshipStruct rss[1];
	rss[0].TargetRange.targetRangeStart = 2.0;
	rss[0].TargetRange.targetRangeEnd = 4.0;
	rss[0].AngleRange.targetRangeStart = 0.01*PI;
	rss[0].AngleRange.targetRangeEnd = PI;
	rss[0].DegreesOfAtrraction = 2.0;
	rss[0].SourceIndex = 0;
	rss[0].TargetIndex = 1;

	//for (int i = 0; i < NRel; i++) {
	//	rss[i].TargetRange.targetRangeStart = 0.0;
	//	rss[i].TargetRange.targetRangeEnd = 2.0;
	//	rss[i].Source.x = 0.0 + i;
	//	rss[i].Source.y = 0.0 + i;
	//	rss[i].Source.z = 0.0;
	//	rss[i].Source.rotX = 1.0;
	//	rss[i].Source.rotY = 1.0;
	//	rss[i].Source.rotZ = 1.0;
	//	rss[i].Target.x = 3.0 + i;
	//	rss[i].Target.y = 3.0 + i;
	//	rss[i].Target.z = 0.0;
	//	rss[i].Target.rotX = 1.0;
	//	rss[i].Target.rotY = 1.0;
	//	rss[i].Target.rotZ = 1.0;
	//	rss[i].DegreesOfAtrraction = 2.0;
	//}

	// Point test code:

	result *result = KernelWrapper(rss, cfg, clearances, offlimits, vtx, surfaceRectangle, &srf, &gpuCfg);
	printf("Results:\n");
	for (int i = 0; i < gpuCfg.gridxDim; i++)
	{
		printf("Result %d\n", i);
		for (int j = 0; j < srf.nObjs; j++) {
			printf("Point [%d] X,Y,Z: %f, %f, %f	Rotation: %f, %f, %f\n", 
				j,
				result[i].points[j].x, 
				result[i].points[j].y, 
				result[i].points[j].z, 
				result[i].points[j].rotX, 
				result[i].points[j].rotY,
				result[i].points[j].rotZ);
		}
		
	}
 	return EXIT_SUCCESS;
}